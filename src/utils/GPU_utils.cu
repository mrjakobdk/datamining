#include "hip/hip_runtime.h"
//
// Created by mrjak on 20-07-2021.
//

#include "GPU_utils.cuh"
#include <cstdio>
#include <map>

#define SECTION_SIZE 128
#define BLOCK_SIZE 512

__global__ void scan_kernel_eff(int *x, int *y, int n) {
    /**
 * from the cuda book
 */
    __shared__ int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }
}

__global__ void scan_kernel_eff(unsigned int *x, unsigned int *y, int n) {
    /**
 * from the cuda book
 */
    __shared__ unsigned int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }
}

__global__ void scan_kernel_eff_large1(int *x, int *y, int *S, int n) {
    /**
 * from the cuda book
 */
    __shared__ int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }

    if (threadIdx.x == 0) {
        S[blockIdx.x] = XY[SECTION_SIZE - 1];
    }
}

__global__ void scan_kernel_eff_large1(unsigned int *x, unsigned int *y, unsigned int *S, int n) {
    /**
 * from the cuda book
 */
    __shared__ unsigned int XY[SECTION_SIZE];

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        XY[threadIdx.x] = x[i];
    }

    for (unsigned int stride = 1; stride <= blockDim.x; stride *= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * 2 * stride - 1;
        if (index < blockDim.x) {
            XY[index] += XY[index - stride];
        }
    }

    for (int stride = SECTION_SIZE; stride > 0; stride /= 2) {
        __syncthreads();
        int index = (threadIdx.x + 1) * stride * 2 - 1;
        if (index + stride < SECTION_SIZE) {
            XY[index + stride] += XY[index];
        }
    }

    __syncthreads();

    if (i < n) {
        y[i] = XY[threadIdx.x];
    }

    if (threadIdx.x == 0) {
        S[blockIdx.x] = XY[SECTION_SIZE - 1];
    }
}

__global__ void scan_kernel_eff_large3(int *y, int *S, int n) {
    /**
 * from the cuda book
 */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x > 0 && i < n) {
        y[i] += S[blockIdx.x - 1];
    }
}

__global__ void scan_kernel_eff_large3(unsigned int *y, unsigned int *S, int n) {
    /**
 * from the cuda book
 */
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (blockIdx.x > 0 && i < n) {
        y[i] += S[blockIdx.x - 1];
    }
}

size_t total_memory_usage = 0;
size_t max_memory_usage = 0;
std::map<void*, size_t> memory_usage_map;

void gpu_malloc(void **devPtr, size_t size) {
    hipMalloc(devPtr, size);
    total_memory_usage += size;
    memory_usage_map[devPtr[0]] = size;
    if (total_memory_usage > max_memory_usage) {
        max_memory_usage = total_memory_usage;
    }
}

void gpu_free(void *devPtr){
    total_memory_usage -= memory_usage_map[devPtr];
    hipFree(devPtr);
}

size_t gpu_max_memory_usage(){
    return max_memory_usage;
}

void gpu_reset_max_memory_usage(){
    max_memory_usage = 0;
}

size_t gpu_total_memory_usage(){
    return total_memory_usage;
}

void inclusive_scan(int *source, int *result, int n) {
    int numBlocks = n / SECTION_SIZE;
    if (n % SECTION_SIZE)
        numBlocks++;

    if (n > SECTION_SIZE) {
        int *S;
        gpu_malloc((void **) &S, numBlocks * sizeof(int));
        scan_kernel_eff_large1<<<numBlocks, SECTION_SIZE>>>(source, result, S, n);
        inclusive_scan(S, S, numBlocks);
        scan_kernel_eff_large3<<<numBlocks, SECTION_SIZE>>>(result, S, n);
        gpu_free(S);
    } else {
        scan_kernel_eff<<<numBlocks, SECTION_SIZE>>>(source, result, n);
    }
}

void inclusive_scan(unsigned int *source, unsigned int *result, int n) {
    int numBlocks = n / SECTION_SIZE;
    if (n % SECTION_SIZE)
        numBlocks++;

    if (n > SECTION_SIZE) {
        unsigned int *S;
        gpu_malloc((void **) &S, numBlocks * sizeof(unsigned int));
        scan_kernel_eff_large1<<<numBlocks, SECTION_SIZE>>>(source, result, S, n);
        inclusive_scan(S, S, numBlocks);
        scan_kernel_eff_large3<<<numBlocks, SECTION_SIZE>>>(result, S, n);
        gpu_free(S);
    } else {
        scan_kernel_eff<<<numBlocks, SECTION_SIZE>>>(source, result, n);
    }
}

int *gpu_malloc_int(int n) {
    if (n <= 0)
        return nullptr;
    int *tmp;
    gpu_malloc((void**)&tmp, n * sizeof(int));
    return tmp;
}

unsigned int *gpu_malloc_unsigned_int(int n) {
    if (n <= 0)
        return nullptr;
    unsigned int *tmp;
    gpu_malloc((void**)&tmp, n * sizeof(unsigned int));
    return tmp;
}

float *gpu_malloc_float(int n) {
    if (n <= 0)
        return nullptr;
    float *tmp;
    gpu_malloc((void**)&tmp, n * sizeof(float));
    return tmp;
}

bool *gpu_malloc_bool(int n) {
    if (n <= 0)
        return nullptr;
    bool *tmp;
    gpu_malloc((void**)&tmp, n * sizeof(bool));
    return tmp;
}

int *gpu_malloc_int_zero(int n) {
    if (n <= 0)
        return nullptr;
    int *tmp;
    gpu_malloc((void**)&tmp, n * sizeof(int));
    hipMemset(tmp, 0, n * sizeof(int));
    return tmp;
}

float *gpu_malloc_float_zero(int n) {
    if (n <= 0)
        return nullptr;
    float *tmp;
    gpu_malloc((void**)&tmp, n * sizeof(float));
    hipMemset(tmp, 0, n * sizeof(float));
    return tmp;
}

bool *gpu_malloc_bool_false(int n) {
    if (n <= 0)
        return nullptr;
    bool *tmp;
    gpu_malloc((void**)&tmp, n * sizeof(bool));
    hipMemset(tmp, 0, n * sizeof(bool));
    return tmp;
}

void copy_D_to_H(int *h_out, int *d_in, int n) {
    hipMemcpy(h_out, d_in, n * sizeof(int), hipMemcpyDeviceToHost);
}

void copy_D_to_H(unsigned int *h_out, unsigned int *d_in, int n) {
    hipMemcpy(h_out, d_in, n * sizeof(unsigned int), hipMemcpyDeviceToHost);
}

void copy_D_to_H(float *h_out, float *d_in, int n) {
    hipMemcpy(h_out, d_in, n * sizeof(float), hipMemcpyDeviceToHost);
}

void copy_D_to_H(bool *h_out, bool *d_in, int n) {
    hipMemcpy(h_out, d_in, n * sizeof(bool), hipMemcpyDeviceToHost);
}

int *copy_D_to_H(int *d_array, int n) {
    int *h_array = new int[n];
    hipMemcpy(h_array, d_array, n * sizeof(int), hipMemcpyDeviceToHost);
    return h_array;
}

float *copy_D_to_H(float *d_array, int n) {
    float *h_array = new float[n];
    hipMemcpy(h_array, d_array, n * sizeof(float), hipMemcpyDeviceToHost);
    return h_array;
}

bool *copy_D_to_H(bool *d_array, int n) {
    bool *h_array = new bool[n];
    hipMemcpy(h_array, d_array, n * sizeof(bool), hipMemcpyDeviceToHost);
    return h_array;
}

int *copy_H_to_D(int *h_array, int n) {
    int *d_array = gpu_malloc_int(n);
    hipMemcpy(d_array, h_array, n * sizeof(int), hipMemcpyHostToDevice);
    return d_array;
}

float *copy_H_to_D(float *h_array, int n) {
    float *d_array = gpu_malloc_float(n);
    hipMemcpy(d_array, h_array, n * sizeof(float), hipMemcpyHostToDevice);
    return d_array;
}

bool *copy_H_to_D(bool *h_array, int n) {
    bool *d_array = gpu_malloc_bool(n);
    hipMemcpy(d_array, h_array, n * sizeof(bool), hipMemcpyHostToDevice);
    return d_array;
}

float *copy_D_to_D(float *d_array1, int n) {
    float *d_array2 = gpu_malloc_float(n);
    hipMemcpy(d_array2, d_array1, n * sizeof(float), hipMemcpyDeviceToDevice);
    return d_array2;
}

void copy_H_to_D(int *d_out, int *h_in, int n) {
    hipMemcpy(d_out, h_in, n * sizeof(int), hipMemcpyHostToDevice);
}

void copy_H_to_D(float *d_out, float *h_in, int n) {
    hipMemcpy(d_out, h_in, n * sizeof(float), hipMemcpyHostToDevice);
}

void copy_H_to_D(bool *d_out, bool *h_in, int n) {
    hipMemcpy(d_out, h_in, n * sizeof(bool), hipMemcpyHostToDevice);
}

void copy_D_to_D(int *d_out, int *d_in, int n) {
    hipMemcpy(d_out, d_in, n * sizeof(int), hipMemcpyDeviceToDevice);
}

void copy_D_to_D(float *d_out, float *d_in, int n) {
    hipMemcpy(d_out, d_in, n * sizeof(float), hipMemcpyDeviceToDevice);
}

void copy_D_to_D(bool *d_out, bool *d_in, int n) {
    hipMemcpy(d_out, d_in, n * sizeof(bool), hipMemcpyDeviceToDevice);
}

int copy_last_D_to_H(int *d_array, int n) {
    int tmp = 0;

    if (n > 0) {
        copy_D_to_H(&tmp, &d_array[n - 1], 1);
    }

    return tmp;
}

int copy_last_D_to_H(unsigned int *d_array, int n) {
    unsigned int tmp = 0;

    if (n > 0) {
        copy_D_to_H(&tmp, &d_array[n - 1], 1);
    }

    return tmp;
}

float copy_last_D_to_H(float *d_array, int n) {
    float tmp = 0.;

    if (n > 0) {
        copy_D_to_H(&tmp, &d_array[n - 1], 1);
    }

    return tmp;
}

void gpu_set_all_zero(int *d_var, int n) {
    if (n > 0) {
        hipMemset(d_var, 0, n * sizeof(int));
    }
}

void gpu_set_all_zero(unsigned int *d_var, int n) {
    if (n > 0) {
        hipMemset(d_var, 0, n * sizeof(unsigned int));
    }
}

void gpu_set_all_zero(float *d_var, int n) {
    if (n > 0) {
        hipMemset(d_var, 0, n * sizeof(float));
    }
}

__global__ void gpu_malloc_int_set_value_kernel(int *d_temp, int size, int value) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < size) {
        d_temp[idx] = value;
    }
}

void gpu_set_all(int *d_var, int size, int value) {
    int numBlocks = size / BLOCK_SIZE;
    if (size % BLOCK_SIZE) {
        numBlocks++;
    }
    gpu_malloc_int_set_value_kernel<<<numBlocks, min(size, BLOCK_SIZE)>>>(d_var, size, value);
}

__global__ void gpu_malloc_int_set_value_kernel(float *d_temp, int size, float value) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;

    if (idx < size) {
        d_temp[idx] = value;
    }
}

void gpu_set_all(float *d_var, int size, float value) {
    int numBlocks = size / BLOCK_SIZE;
    if (size % BLOCK_SIZE) {
        numBlocks++;
    }
    gpu_malloc_int_set_value_kernel<<<numBlocks, min(size, BLOCK_SIZE)>>>(d_var, size, value);
}

__global__ void print_array_gpu_kernel(int *x, int n) {
    for (int i = 0; i < n; i++) {
        if (x[i] < 10 && x[i] > -1)
            printf(" ");
        if (x[i] < 100 && x[i] > -10)
            printf(" ");
        printf("%d ", x[i]);
    }
    printf("\n");
}

void print_array_gpu(int *x, int n) {
    print_array_gpu_kernel<<<1, 1>>>(x, n);
    hipDeviceSynchronize();
}

__global__ void print_array_nonzero_gpu_kernel(int *x, int n) {
    for (int i = 0; i < n; i++) {
        if (x[i] < 10 && x[i] > -1)
            printf(" ");
        if (x[i] < 100 && x[i] > -10)
            printf(" ");
        if (x[i] < 1000 && x[i] > -100)
            printf(" ");
        if (x[i] != 0) {
            printf("%d ", x[i]);
        } else {
            printf("  ");
        }
    }
    printf("\n");
}

void print_array_nonzero_gpu(int *x, int n) {
    print_array_nonzero_gpu_kernel<<<1, 1>>>(x, n);
    hipDeviceSynchronize();
}

__global__ void print_array_gpu_kernel(float *x, int n) {
    for (int i = 0; i < n; i++) {
        printf("%f ", x[i]);
    }
    printf("\n");
}

void print_array_gpu(float *x, int n) {
    print_array_gpu_kernel<<<1, 1>>>(x, n);
    hipDeviceSynchronize();
}

__global__ void print_array_gpu_kernel(bool *x, int n) {
    for (int i = 0; i < n; i++) {
        if (x[i])
            printf("1 ");
        else
            printf("0 ");
    }
    printf("\n");
}

void print_array_gpu(bool *x, int n) {
    print_array_gpu_kernel<<<1, 1>>>(x, n);
    hipDeviceSynchronize();
}

__global__ void print_array_gpu_kernel(float *x, int n, int m) {
    for (int i = 0; i < n * m; i++) {
        if (x[i] < 10)
            printf(" ");
        if (x[i] < 100)
            printf(" ");
        printf("%f ", (float) x[i]);
        if ((i + 1) % m == 0) {
            printf("\n");
        }
    }
    printf("\n");
}

void print_array_gpu(float *d_X, int n, int m) {
    print_array_gpu_kernel<<<1, 1>>>(d_X, n, m);
    hipDeviceSynchronize();
}

__global__ void print_array_gpu_kernel(int *x, int n, int m) {
    for (int i = 0; i < n * m; i++) {
        if (x[i] < 10)
            printf(" ");
        if (x[i] < 100)
            printf(" ");
        printf("%d ", x[i]);
        if ((i + 1) % m == 0) {
            printf("\n");
        }
    }
    printf("\n");
}

void print_array_gpu(int *d_X, int n, int m) {
    print_array_gpu_kernel<<<1, 1>>>(d_X, n, m);
    hipDeviceSynchronize();
}


__global__
void print_array_gpu_kernel(bool *x, int n, int m) {
    for (int i = 0; i < n * m; i++) {
        if (x[i]) {
            printf("true ");
        } else {
            printf("false ");
        }
        if ((i + 1) % m == 0) {
            printf("\n");
        }
    }
    printf("\n");
}


void print_array_gpu(bool *d_X, int n, int m) {
    print_array_gpu_kernel << < 1, 1 >> > (d_X, n, m);
    hipDeviceSynchronize();
}

__device__ int get_start(const int *d_array, const int idx) {
    return idx > 0 ? d_array[idx - 1] : 0;
}

__device__ int get_end(const int *d_array, const int idx) {
    return d_array[idx];
}


__global__
void set_kernel(int *x, int i, int value) {
    x[i] = value;
}

void set(int *x, int i, int value) {
    set_kernel << < 1, 1 >> > (x, i, value);
}


__global__
void set_kernel(float *x, int i, float value) {
    x[i] = value;
}

void set(float *x, int i, float value) {
    set_kernel << < 1, 1 >> > (x, i, value);
}


__global__
void set_kernel(int *x, int *idx, int i, int value) {
    x[i] = idx[value];
}

void set(int *x, int *idx, int i, int value) {
    set_kernel << < 1, 1 >> > (x, idx, i, value);
}


__global__
void set_all(float *d_X, float value, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        d_X[i] = value;
    }
}



__global__
void set_all(int *d_X, int value, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        d_X[i] = value;
    }
}

__global__
void set_all(bool *d_X, bool value, int n) {
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x) {
        d_X[i] = value;
    }
}

__global__
void init_seed(hiprandState *state, int seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(seed, idx, 0, &state[idx]);
}



__global__ void gpu_random_sample_kernel_locked_v2(int *d_in, int k, int n, hiprandState *state, int *d_lock) {
    for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < k; i += gridDim.x * blockDim.x) {

        int j = hiprand(&state[threadIdx.x]) % (n);

        if (i > j) {
            int tmp = j;
            j = i;
            i = tmp;
        }

        bool success = false;

        while (!success) {
            if (atomicCAS(&d_lock[i], 0, 1) == 0) {
                if (i == j || atomicCAS(&d_lock[j], 0, 1) == 0) {
                    int tmp_idx = d_in[i];
                    d_in[i] = d_in[j];
                    d_in[j] = tmp_idx;

                    success = true;
                    if (i != j) {
                        atomicExch(&d_lock[j], 0);
                    }
                }
                atomicExch(&d_lock[i], 0);
            }
        }
    }
}

void gpu_random_sample_locked(int *d_in, int k, int n, hiprandState *d_state, int *d_lock) {
    hipMemset(d_lock, 0, n * sizeof(int));
    int number_of_blocks = n / BLOCK_SIZE;
    if (n % BLOCK_SIZE) number_of_blocks++;
    gpu_random_sample_kernel_locked_v2 << < number_of_blocks, min(k, BLOCK_SIZE) >> > (d_in, k, n, d_state, d_lock);
}


__global__
void gpu_gather_1d_kernel(int *d_source, int *d_indices, int length,
                          int *d_result) {//todo change order
    for (int j = 0; j < length; j++) {
        d_result[j] = d_source[d_indices[j]];
    }
}

void gpu_gather_1d(int *d_result, int *d_source, int *d_indices, int length) {
    gpu_gather_1d_kernel << < 1, 1 >> > (d_source, d_indices, length, d_result);
}