#include "hip/hip_runtime.h"
//
// Created by jakobrj on 3/8/21.
//

#include "GPU_PROCLUS.cuh"
#include "../../utils/GPU_utils.cuh"

#import <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

#define BLOCK_SIZE 1024
#define BLOCK_SIZE_SMALL 128

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int get_current_memory_usage() {
    size_t free_byte;
    size_t total_byte;
    hipMemGetInfo(&free_byte, &total_byte);
    return total_byte - free_byte;
}

__device__ __forceinline__

float atomicMin(float *addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMin((int *) addr, __float_as_int(value))) :
          __uint_as_float(atomicMax((unsigned int *) addr, __float_as_uint(value)));

    return old;
}

__device__ __forceinline__

float atomicMax(float *addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *) addr, __float_as_int(value))) :
          __uint_as_float(atomicMin((unsigned int *) addr, __float_as_uint(value)));

    return old;
}

__global__
void gpu_greedy_kernel_dist_max(float *d_max_value, float *d_data, int *M, int *d_S, float *dist, int Ak, int d,
                                int mediod_idx) {
    int m_i = M[mediod_idx];
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < Ak; i += blockDim.x * gridDim.x) {
        float distance = 0;
        for (int j = 0; j < d; j++) {
            float sub = d_data[d_S[i] * d + j] - d_data[m_i * d + j];
            distance += sub * sub;
        }
        dist[i] = sqrt(distance);
    }

    __shared__ float max_value;
    max_value = 0.;
    __syncthreads();

    for (int v = blockIdx.x * blockDim.x + threadIdx.x; v < Ak; v += blockDim.x * gridDim.x) {
        atomicMax(&max_value, dist[v]);//todo does this work?
    }

    __syncthreads();
    if (threadIdx.x == 0)
        atomicMax(&d_max_value[0], max_value);
}

__global__
void gpu_greedy_kernel_largest_2(float *d_max_value, int *d_S, int *M, float *dist, int *d_prev, int Ak, int i, int n) {
    for (int v = blockIdx.x * blockDim.x + threadIdx.x; v < Ak; v += blockDim.x * gridDim.x) {
        if (dist[v] == d_max_value[0]) {
            M[i] = d_S[v];
            d_prev[0] = v;
        }
    }
}

__global__
void gpu_greedy_kernel_dist_min_max(float *d_max_value, float *d_data, int *M, int *d_S, float *dist, int Ak, int d,
                                    int mediod_idx) {

    __shared__ float max_value;
    max_value = 0.;
    __syncthreads();

    int m_i = M[mediod_idx];
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < Ak; i += blockDim.x * gridDim.x) {
        float distance = 0;
        for (int j = 0; j < d; j++) {
            float sub = d_data[d_S[i] * d + j] - d_data[m_i * d + j];
            distance += sub * sub;
        }

        distance = sqrt(distance);
        //finding the min distance here instead of in two kernels reduced the running time by 4ms out of 20ms
        if (distance < dist[i]) {
            dist[i] = distance;
        }

        atomicMax(&max_value, dist[i]);//first find local (within block)
    }


    __syncthreads();
    if (threadIdx.x == 0)
        atomicMax(&d_max_value[0], max_value);//then find global
}

int *gpu_greedy(float *d_data, int *d_S,
                int *d_M, float *d_dist, int *d_prev, float *d_max_value,
                int Bk, int Ak, int d, int n) {


    //hipMalloc(&d_M, Bk * sizeof(int));
    //hipMalloc(&d_dist, Ak * sizeof(float));
    //hipMalloc(&d_prev, sizeof(int));
    //hipMemset(d_prev, 0, sizeof(int));
    //hipMalloc(&d_max_value, sizeof(float));
    //hipMemset(d_max_value, 0, sizeof(float));

//    int rnd_start = std::rand() % Ak;
    int rnd_start = Ak / 2;

    set(d_M, d_S, 0, rnd_start);
    int number_of_blocks = Ak / BLOCK_SIZE_SMALL;
    if (Ak % BLOCK_SIZE_SMALL) number_of_blocks++;
    dim3 grid(number_of_blocks);

    gpu_greedy_kernel_dist_max << < grid, BLOCK_SIZE_SMALL >> > (d_max_value, d_data, d_M, d_S, d_dist, Ak, d, 0);

    for (int i = 1; i < Bk; i++) {

        gpu_greedy_kernel_largest_2 << < grid, BLOCK_SIZE_SMALL >> > (d_max_value, d_S, d_M, d_dist, d_prev, Ak, i, n);

        hipMemset(d_max_value, 0, sizeof(float));

        gpu_greedy_kernel_dist_min_max << < grid, BLOCK_SIZE_SMALL >> >
        (d_max_value, d_data, d_M, d_S, d_dist, Ak, d, i);
    }

    return d_M;
}

__global__
void gpu_compute_L_kernel_sum_dist_V2(float *d_dist_n_k, int *d_M_current, float *d_data, int n, int d, int k) {
    int i = blockIdx.x;
    int m_i = d_M_current[i];

    extern __shared__ float s_data_i[];

    if (threadIdx.x < d) {
        s_data_i[threadIdx.x] = d_data[m_i * d + threadIdx.x];
    }

    __syncthreads();

    for (int p = blockIdx.y * blockDim.x + threadIdx.x; p < n; p += gridDim.y * blockDim.x) {//independent
        float sum = 0;
        for (int j = 0; j < d; j++) {//we have plenty to parallelize over here - so we can avoid the atomic
            float sub = d_data[p * d + j] - s_data_i[j];
            sum += sub * sub;
        }
        d_dist_n_k[i * n + p] = std::sqrt(sum);
    }
}

__global__
void gpu_compute_L_kernel_compute_delta_V2(float *d_delta, float *d_dist_n_k, int *d_M_current, int n, int k) {
    for (int i = threadIdx.x; i < k; i += blockDim.x) {//independent
        d_delta[i] = 1000000.;//todo not nice
        for (int j = 0; j < k; j++) {
            int p = d_M_current[j];
            if (i != j) {
                if (d_dist_n_k[i * n + p] <= d_delta[i]) {
                    d_delta[i] = d_dist_n_k[i * n + p];
                }
            }
        }
    }
}

__global__
void gpu_compute_L_kernel_compute_L_V2(int *d_L, int *d_L_sizes, float *d_delta, float *d_dist_n_k, int n, int k) {
    for (int i = blockIdx.x; i < k; i += gridDim.x) {//independent
        for (int p = threadIdx.x; p < n; p += blockDim.x) {
            if (d_dist_n_k[i * n + p] <= d_delta[i]) {
                int old_size = atomicInc((unsigned int *) &d_L_sizes[i], n);
                d_L[i * n + old_size] = p;
            }
        }
    }
}

void gpu_compute_L(int *d_L, int *d_L_sizes,
                   float *d_dist_n_k,
                   float *d_delta,
                   int *d_M_current,
                   float *d_data,
                   int n, int d, int k) {

    int number_of_blocks = n / BLOCK_SIZE_SMALL;
    if (n % BLOCK_SIZE_SMALL) number_of_blocks++;
    dim3 grid_k_n(k, number_of_blocks);
    gpu_compute_L_kernel_sum_dist_V2 << < grid_k_n, min(n, BLOCK_SIZE_SMALL), d * sizeof(float) >> >
    (d_dist_n_k, d_M_current,
            d_data,
            n, d, k);

    //compute delta
    gpu_compute_L_kernel_compute_delta_V2 << < 1, k >> > (d_delta, d_dist_n_k, d_M_current, n, k);

    //compute L
    hipMemset(d_L_sizes, 0, k * sizeof(int));
    gpu_compute_L_kernel_compute_L_V2 << < k, min(n, BLOCK_SIZE) >> > (d_L, d_L_sizes, d_delta, d_dist_n_k, n, k);
}

__global__
void
gpu_find_dimensions_kernel_Z(float *__restrict__ d_Z, const float *__restrict__ d_X, const int k, const int d) {

    int i = blockIdx.x;//independent for different k
    int j = threadIdx.x;//independent for different d

    __shared__ float Y_i;
    Y_i = 0.;
    __shared__ float sigma_i;
    sigma_i = 0.;
    __syncthreads();

    float X_ij = d_X[i * d + j];
    atomicAdd(&Y_i, X_ij / d);
    __syncthreads();
////
    float sub = X_ij - Y_i;
    atomicAdd(&sigma_i, sub * sub);
    __syncthreads();
    if (threadIdx.x == 0) {//only one should do this
        sigma_i /= (d - 1);
        sigma_i = std::sqrt(sigma_i);
    }
    __syncthreads();
////
    d_Z[i * d + j] = sub / sigma_i;
}

__global__
void gpu_find_dimensions_kernel_X(float *d_X,
                                  int *d_L, int *d_L_sizes,
                                  int *d_M_current,
                                  float *d_data,
                                  int n, int d, int k) {
    int i = blockIdx.x; //independent for different k
    int j = threadIdx.x; //independent for different d

    int m_i = d_M_current[i];
    int L_i_sizes = d_L_sizes[i];
    float data_ij = d_data[m_i * d + j];

    float sum = 0.;
    for (int p = blockDim.y * blockIdx.y + threadIdx.y; p < L_i_sizes; p += gridDim.y * blockDim.y) {
        int point = d_L[i * n + p];
        sum += std::abs(d_data[point * d + j] - data_ij);
    }

    atomicAdd(&d_X[i * d + j], sum / L_i_sizes);

}


__global__
void gpu_find_dimensions_kernel_X_v2(float *d_X,
                                     int *d_L, int *d_L_sizes,
                                     int *d_M_current,
                                     float *d_data,
                                     int n, int d, int k) {
    int i = blockIdx.x; //independent for different k
    int j = blockIdx.y; //independent for different d

    int m_i = d_M_current[i];
    int L_i_sizes = d_L_sizes[i];
    float data_ij = d_data[m_i * d + j];

    float sum = 0.;
    for (int p = threadIdx.x; p < L_i_sizes; p += blockDim.x) {
        int point = d_L[i * n + p];
        sum += std::abs(d_data[point * d + j] - data_ij);
    }

    atomicAdd(&d_X[i * d + j], sum / L_i_sizes);

}


__global__
void gpu_find_dimensions_kernel_compute_D(int *d_D, float *d_Z, int k, int d, int l) {
    //# ensuring that we find atleast 2 for each and than the k*l #todo fast - sort first instead

    extern __shared__ float min_values[];
    int *i_was_firsts = (int *) &min_values[k];
    __shared__ float min_value;
    __shared__ int i_was_first;

    for (int _ = 0; _ < 2; _++) {

        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            min_values[i] = 1000000.;//todo not nice
            i_was_firsts[i] = 1;
        }
        __syncthreads();

        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            for (int j = threadIdx.y; j < d; j += blockDim.y) {
                atomicMin(&min_values[i], d_Z[i * d + j]);
            }
        }
        __syncthreads();

        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            for (int j = threadIdx.y; j < d; j += blockDim.y) {
                if (d_Z[i * d + j] == min_values[i]) {
                    int was_i_first = atomicCAS(&i_was_firsts[i], 1, 0); //(old == compare ? val : old)
                    if (was_i_first) {
                        d_Z[i * d + j] = 1000000.;//todo not nice
                        d_D[i * d + j] = 1;
                    }
                }
            }
        }
        __syncthreads();
    }

    for (int _ = k * 2; _ < k * l; _++) {
        min_value = 1000000.;//todo not nice
        i_was_first = 1;
        __syncthreads();

        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            for (int j = threadIdx.y; j < d; j += blockDim.y) {
                atomicMin(&min_value, d_Z[i * d + j]);
            }
        }

        __syncthreads();
        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            for (int j = threadIdx.y; j < d; j += blockDim.y) {
                if (d_Z[i * d + j] == min_value) {
                    int was_i_first = atomicCAS(&i_was_first, 1, 0); //(old == compare ? val : old)
                    if (was_i_first) {
                        d_Z[i * d + j] = 1000000.;//todo not nice
                        d_D[i * d + j] = 1;
                    }
                }
            }
        }
        __syncthreads();
    }
}


__global__
void gpu_find_dimensions_kernel_D(int *d_D, float *d_Z, int k, int d, int l) {
    //# ensuring that we find atleast 2 for each and than the k*l #todo fast - sort first instead

    for (int _ = 0; _ < 2; _++) {
        for (int i = 0; i < k; i++) {
            float min_value = 100000.;
            int best_j = 0;
            for (int j = 0; j < d; j++) {
                if (d_Z[i * d + j] < min_value) {
                    min_value = d_Z[i * d + j];
                    best_j = j;;
                }
            }
            d_Z[i * d + best_j] = 1000000.;//todo not nice
            d_D[i * d + best_j] = 1;
        }
    }

    for (int _ = k * 2; _ < k * l; _++) {
        float min_value = 100000.;
        int best_i = 0;
        int best_j = 0;
        for (int i = 0; i < k; i++) {
            for (int j = 0; j < d; j++) {
                if (d_Z[i * d + j] < min_value) {
                    min_value = d_Z[i * d + j];
                    best_j = j;
                    best_i = i;
                }
            }
        }
        d_Z[best_i * d + best_j] = 1000000.;//todo not nice
        d_D[best_i * d + best_j] = 1;
    }
}

void gpu_find_dimensions(int *d_D, float *d_Z, float *d_X,
                         int *d_L, int *d_L_sizes,
                         int *d_M_current,
                         float *d_data,
                         int n, int d, int k, int l) {
    int number_of_blocks = (k * d) / BLOCK_SIZE;
    if ((k * d) % BLOCK_SIZE) number_of_blocks++;

    set_all << < number_of_blocks, min(k * d, BLOCK_SIZE) >> > (d_X, 0, k * d);


    int remaining_d = BLOCK_SIZE / d;
    int number_of_blocks_X_join_v2 = (n / k) / remaining_d;
    if ((n / k) % remaining_d) number_of_blocks_X_join_v2++;
    dim3 grid_X_join_v2(k, number_of_blocks_X_join_v2);
    dim3 block_X_join_v2(d, remaining_d);
//    dim3 grid_X_join_v2(k, d);
//    dim3
//    block_X_join_v2(BLOCK_SIZE);

    hipMemset(d_X, 0, d * k * sizeof(float));
    gpu_find_dimensions_kernel_X << < grid_X_join_v2, block_X_join_v2 >> > (d_X,
            d_L, d_L_sizes,
            d_M_current,
            d_data,
            n, d, k);
//    gpu_find_dimensions_kernel_X_v2 << < grid_X_join_v2, block_X_join_v2 >> > (d_X,
//            d_L, d_L_sizes,
//            d_M_current,
//            d_data,
//            n, d, k);

    gpu_find_dimensions_kernel_Z << < k, d >> > (d_Z, d_X, k, d);

    //compute D
    set_all << < number_of_blocks, min(k * d, BLOCK_SIZE) >> > (d_D, false, k * d);
    dim3 block(min(32, k), min(32, d));
    gpu_find_dimensions_kernel_compute_D << < 1, block, 2 * k * sizeof(float) >> > (d_D, d_Z, k, d, l);
}

__global__
void
gpu_restructure_D(int *__restrict__ d_Ds, int *__restrict__ d_D_sizes, const int *__restrict__ d_D, int d, int k) {

    int i = blockIdx.x;
    int j = threadIdx.x;

    if (d_D[i * d + j]) {
        int idx = atomicInc((unsigned int *) &d_D_sizes[i], d);
        d_Ds[i * d + idx] = j;
    }
}

//__global__
//void
//gpu_assign_points_kernel(int *__restrict__ d_Ds, int *__restrict__ d_D_sizes,
//                         int *__restrict__ d_C, int *__restrict__ d_C_size,
//                         const float *__restrict__ d_data, const int *__restrict__ d_M_current,
//                         const int n, const int k, const int d) {
//
//    for (int p = blockIdx.x * blockDim.x + threadIdx.x; p < n; p += blockDim.x * gridDim.x) {
//        float min_value = 1000000.;
//        int best_i = 0;
//        for (int i = 0; i < k; i++) {
//            int m_i = d_M_current[i];
//            int size = d_D_sizes[i];
//
//            float dist = 0;
//            for (int l = 0; l < size; l++) {
//                int j = d_Ds[i * d + l];
//                dist += abs(d_data[p * d + j] - d_data[m_i * d + j]);
//            }
//            dist /= size;
//
//            if (dist < min_value) {
//                min_value = dist;
//                best_i = i;
//            }
//        }
//        int idx = atomicInc((unsigned int *) &d_C_size[best_i], n);
//        d_C[best_i * n + idx] = p;
//
//    }
//}
__global__
void
gpu_assign_points_kernel(int *__restrict__ d_Ds, int *__restrict__ d_D_sizes,
                         int *__restrict__ d_C, int *__restrict__ d_C_size,
                         const float *__restrict__ d_data, const int *__restrict__ d_M_current,
                         const int n, const int k, const int d) {

    extern __shared__ float s_min_value[];

    float dist = 0;

    int i = threadIdx.y;
    int m_i = d_M_current[i];
    int size = d_D_sizes[i];

    int p = blockIdx.x * blockDim.x + threadIdx.x;

    s_min_value[threadIdx.x] = 1000000.;
    __syncthreads();

    if (p < n) {
        dist = 0;

        for (int l = 0; l < size; l++) {
            int j = d_Ds[i * d + l];
            dist += abs(d_data[p * d + j] - d_data[m_i * d + j]);
        }

        dist /= size;

        atomicMin(&s_min_value[threadIdx.x], dist);
    }

    __syncthreads();

    if (p < n) {
        if (dist == s_min_value[threadIdx.x]) {
            int idx = atomicInc((unsigned int *) &d_C_size[i], n);
            d_C[i * n + idx] = p;
        }
    }
}

/*
__global__
void
gpu_assign_points_kernel_CG(int *__restrict__ d_Ds, int *__restrict__ d_D_sizes,
                            int *__restrict__ d_C, int *__restrict__ d_C_size,
                            const float *__restrict__ d_data, const int *__restrict__ d_M_current,
                            const int n, const int k, const int d) {

    extern __shared__ float s_min_value[];

    int point_group = threadIdx.y;

    cg::coalesced_group active = cg::coalesced_threads();
    cg::coalesced_group subgroup = cg::labeled_partition(active, point_group);

    float dist = 0;

    int i = threadIdx.x;
    int m_i = d_M_current[i];
    int size = d_D_sizes[i];

    int p = blockIdx.x * blockDim.y + threadIdx.y;

    s_min_value[point_group] = 1000000.;
    subgroup.sync();

    if (p < n) {
        dist = 0;

        for (int l = 0; l < size; l++) {
            int j = d_Ds[i * d + l];
            dist += abs(d_data[p * d + j] - d_data[m_i * d + j]);
        }

        dist /= size;

        atomicMin(&s_min_value[point_group], dist);
    }

    subgroup.sync();

    if (p < n) {
        if (dist == s_min_value[point_group]) {
            int idx = atomicInc((unsigned int *) &d_C_size[i], n);
            d_C[i * n + idx] = p;
        }
    }
}
*/

void gpu_assign_points(int *d_C, int *d_C_sizes,
                       int *d_D, int *d_Ds, int *d_D_sizes,
                       int *d_M_current,
                       float *d_data,
                       int n, int d, int k) {

    int remaining = BLOCK_SIZE_SMALL / k;
    int number_of_blocks = n / remaining;
    if (n % remaining) number_of_blocks++;
    dim3 block_n_k(min(n, remaining), k);
    gpuErrchk(hipPeekAtLastError());

//    int number_of_blocks = n / BLOCK_SIZE;
//    if (n % BLOCK_SIZE) number_of_blocks++;

    hipMemset(d_C_sizes, 0, k * sizeof(float));
    hipMemset(d_Ds, 0, k * d * sizeof(int));
    hipMemset(d_D_sizes, 0, k * sizeof(int));
    gpuErrchk(hipPeekAtLastError());

    gpu_restructure_D << < k, d >> > (d_Ds, d_D_sizes, d_D, d, k);
    gpuErrchk(hipPeekAtLastError());


    gpu_assign_points_kernel << < number_of_blocks, block_n_k, min(n, remaining) * sizeof(float) >> > (
            d_Ds, d_D_sizes, d_C, d_C_sizes, d_data, d_M_current, n, k, d);
    gpuErrchk(hipPeekAtLastError());


/*
    dim3 block_k_n(k, min(n, remaining));
    gpu_assign_points_kernel_CG << < number_of_blocks, block_k_n, min(n, remaining) * sizeof(float) >> > (
            d_Ds, d_D_sizes, d_C, d_C_sizes, d_data, d_M_current, n, k, d);
            */

//    gpu_assign_points_kernel<<<number_of_blocks, BLOCK_SIZE>>>(d_Ds, d_D_sizes, d_C, d_C_sizes, d_data, d_M_current, n,
//                                                               k, d);
}


//todo we should reconsider how we represent clustering - should we represent it in to different way? or just change it at the very end?
__global__
void gpu_evaluate_cluster_kernel(float *d_cost, int *d_C,
                                 int *d_C_size, int *d_D, int *d_D_sizes,
                                 float *d_data,
                                 int n, int d, int k) { //  --  40.99%  413ms

    __shared__ float tmp_mean;
    __shared__ float tmp_cost;
    float tmp;

    int j = blockIdx.x;//j is the dimension within d dimensions
    int i = blockIdx.y;//i is the mediod / cluster within k clusters

    int size = d_C_size[i];
    int tmp_2 = d_D_sizes[i] * n;

    if (d_D[i * d + j]) {
        tmp = 0.;
        tmp_mean = 0.;
        __syncthreads();
        for (int l = threadIdx.x; l < size; l += blockDim.x) {
            int p = d_C[i * n + l];
            tmp += d_data[p * d + j];
        }
        atomicAdd(&tmp_mean, tmp / size);

        tmp_cost = 0;
        __syncthreads();
        tmp = 0.;
        for (int l = threadIdx.x; l < size; l += blockDim.x) {
            int p = d_C[i * n + l];
            tmp += abs(d_data[p * d + j] - tmp_mean);
        }

        atomicAdd(&tmp_cost, tmp / tmp_2);
        __syncthreads();
        if (threadIdx.x == 0)
            atomicAdd(&d_cost[0], tmp_cost);
    }
}

void
gpu_evaluate_cluster(float *d_cost, int *d_C, int *d_C_sizes, int *d_D, int *d_D_sizes, float *d_data,
                     int n, int d, int k) {

    int number_of_blocks = n / BLOCK_SIZE;
    if (n % BLOCK_SIZE) number_of_blocks++;
    dim3 grid(d, k);
    dim3 block(min(BLOCK_SIZE, (int) n / k));

    gpuErrchk(hipPeekAtLastError());
    hipMemset(d_cost, 0, sizeof(float));
    gpuErrchk(hipPeekAtLastError());
    gpu_evaluate_cluster_kernel << < grid, block >> > (d_cost, d_C, d_C_sizes, d_D, d_D_sizes, d_data,
            n, d, k);
    gpuErrchk(hipPeekAtLastError());
}

__global__
void
gpu_update_best_kernel_is_best(float *d_objective_function, float *d_best_objective, int *d_termination_criterion) {
    d_termination_criterion[0]++;
    if (d_objective_function[0] < d_best_objective[0]) {
        d_termination_criterion[0] = 0;
        d_best_objective[0] = d_objective_function[0];
    }
}

__global__
void
gpu_update_best_kernel_init_k(int *d_termination_criterion, int *d_M_best, int *d_M_current,
                              bool *d_bad, int k) {

    if (d_termination_criterion[0] == 0) {//todo worng!!!! then we allways pick the last????
        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            d_M_best[i] = d_M_current[i];
            d_bad[i] = false;
        }
    }
}

__global__
void
gpu_update_best_kernel_C(int *d_C_best, int *d_C_sizes_best, int *d_C, int *d_C_sizes, int *d_termination_criterion,
                         int n) {

    if (d_termination_criterion[0] == 0) {
        int i = blockIdx.x;

        int C_i_size = d_C_sizes[i];
        for (int p_id = threadIdx.x; p_id < C_i_size; p_id += blockDim.x) {
            d_C_best[i * n + p_id] = d_C[i * n + p_id];
        }

        if (threadIdx.x == 0) {
            d_C_sizes_best[i] = d_C_sizes[i];
        }
    }
}

__global__
void gpu_update_best_kernel_find_bad(int *d_C_sizes_best, int *d_termination_criterion, bool *d_bad, int k, int n,
                                     float min_deviation) {

    __shared__ int min_value;
    min_value = 1000000.;

    __syncthreads();

    if (d_termination_criterion[0] == 0) {
        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            atomicMin(&min_value, d_C_sizes_best[i]);
        }
        __syncthreads();

        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            if (d_C_sizes_best[i] == min_value) {
                d_bad[i] = true;
            }
        }
        __syncthreads();

        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            if (d_C_sizes_best[i] < n / k * min_deviation) {
                d_bad[i] = true;
            }
        }
    }
}

void
gpu_update_best(float *d_cost, float *d_cost_best,
                int *d_termination_criterion,
                int *d_M_best, int *d_M_current,
                int *d_C, int *d_C_sizes, int *d_C_best, int *d_C_sizes_best,
                bool *d_bad,
                float min_deviation, int n, int k) {

    gpu_update_best_kernel_is_best << < 1, 1 >> > (d_cost, d_cost_best, d_termination_criterion);
    gpu_update_best_kernel_init_k << < 1, k >> > (d_termination_criterion, d_M_best, d_M_current, d_bad, k);
    gpu_update_best_kernel_C << < k, BLOCK_SIZE >> >
    (d_C_best, d_C_sizes_best, d_C, d_C_sizes, d_termination_criterion, n);
    gpu_update_best_kernel_find_bad << < 1, k >> >
    (d_C_sizes_best, d_termination_criterion, d_bad, k, n, min_deviation);

}

__global__
void gpu_replace_medoids_kernel(int *d_M_current, int *d_M_random, int *d_M, int *d_M_best, bool *d_bad,
                                int k) {

    int j = 0;
    for (int i = 0; i < k; i++) {
        if (!d_bad[i]) {
            d_M_current[j] = d_M_best[i];
            j += 1;
        }
    }

    int p = 0;
    while (j < k) {
        bool is_in = false;
        for (int i = 0; i < j; i++) {
            if (d_M[d_M_random[p]] == d_M_current[i]) {
                is_in = true;
                break;
            }
        }
        if (!is_in) {
            d_M_current[j] = d_M[d_M_random[p]];
            j += 1;
        }
        p += 1;
    }
}

__global__
void remove_outliers_kernel_min_delta(float *d_delta, int *d_D, int *d_M_best, float *d_data, int d, int k) {
    for (int i = blockIdx.x; i < k; i += gridDim.x) {
        for (int j = threadIdx.x; j < k; j += blockDim.x) {
            if (i != j) {
                float msd = 0.;
                int size = 0;
                for (int l = 0; l < d; l++) {//todo could be parallelized
                    if (d_D[i * d + l]) {
                        msd += std::abs(d_data[d_M_best[i] * d + l] - d_data[d_M_best[j] * d + l]);
                        size++;
                    }
                }
                msd /= size;

                atomicMin(&d_delta[i], msd);
            }
        }
    }
}

__global__
void
remove_outliers_kernel_remove(int *d_C_result, int *d_C_best, int *d_C_sizes_best,
                              float *d_delta,
                              int *d_D,
                              int *d_M_best,
                              float *d_data,
                              int n, int d, int k) {

    int i = blockIdx.x;

    int C_i_size = d_C_sizes_best[i];
    for (int p_id = threadIdx.x; p_id < C_i_size; p_id += blockDim.x) {
        int p = d_C_best[i * n + p_id];

        int clustered = -1;
        for (int l = 0; l < k; l++) {

            float msd = 0.;
            int size = 0;
            for (int j = 0; j < d; j++) {
                if (d_D[l * d + j]) {
                    msd += std::abs(d_data[d_M_best[l] * d + j] - d_data[p * d + j]);
                    size++;
                }
            }
            msd /= size;

            if (msd <= d_delta[l]) {
                clustered = i;
                break;
            }
        }

        d_C_result[p] = clustered;

    }
}

void remove_outliers(int *d_C_result, int *d_C_best, int *d_C_sizes_best,
                     int *d_D,
                     float *d_delta,
                     int *d_M_best,
                     float *d_data,
                     int n, int d, int k) {

    set_all << < 1, k >> > (d_delta, 1000000., k);//todo not nice

    remove_outliers_kernel_min_delta << < k, min(k, BLOCK_SIZE) >> > (d_delta,
            d_D,
            d_M_best,
            d_data,
            d, k);

    remove_outliers_kernel_remove << < k, BLOCK_SIZE >> > (d_C_result, d_C_best, d_C_sizes_best,
            d_delta,
            d_D,
            d_M_best,
            d_data,
            n, d, k);

}

__global__
void fill_with_indices_kernel(int *d_S, int n) {
    for (int p = blockIdx.x * blockDim.x + threadIdx.x; p < n; p += blockDim.x * gridDim.x) {
        d_S[p] = p;
    }
}

void fill_with_indices(int *d_S, int n) {
    int number_of_blocks = n / BLOCK_SIZE;
    if (n % BLOCK_SIZE) number_of_blocks++;
    fill_with_indices_kernel << < number_of_blocks, min(n, BLOCK_SIZE) >> > (d_S, n);
}

__global__
void gpu_compute_L_kernel_KEEP_dist(float *d_dist_n_k,
                                    int *d_M_current, int *d_M_bad,
                                    float *d_data,
                                    int n, int d) {
    int i = d_M_bad[blockIdx.x];
    int m_i = d_M_current[i];

    extern __shared__ float s_data_i[];

    if (threadIdx.x < d) {
        s_data_i[threadIdx.x] = d_data[m_i * d + threadIdx.x];
    }

    __syncthreads();

    for (int p = blockIdx.y * blockDim.x + threadIdx.x; p < n; p += gridDim.y * blockDim.x) {//independent
        float sum = 0;
        for (int j = 0; j < d; j++) {//we have plenty to parallelize over here - so we can avoid the atomic
            float sub = d_data[p * d + j] - s_data_i[j];
            sum += sub * sub;
        }
        d_dist_n_k[i * n + p] = std::sqrt(sum);
    }
}

__global__
void gpu_compute_L_kernel_KEEP_L(int *d_lambda, int *d_L, int *d_L_sizes, int *d_L_sizes_change,
                                 float *d_dist_n_k,
                                 float *d_delta, float *d_delta_old,
                                 int n) {
    int i = blockIdx.x;

    if (d_delta_old[i] > d_delta[i]) {
        if (threadIdx.x == 0)
            d_lambda[i] = -1;
        for (int p = threadIdx.x; p < n; p += blockDim.x) {
            if (d_dist_n_k[i * n + p] <= d_delta_old[i] &&
                d_dist_n_k[i * n + p] > d_delta[i]) {
                int idx = atomicInc((unsigned int *) &d_L_sizes_change[i], n);
                d_L[i * n + idx] = p;
            }
        }
    } else {
        if (threadIdx.x == 0)
            d_lambda[i] = 1;
        for (int p = threadIdx.x; p < n; p += blockDim.x) {
            if (d_dist_n_k[i * n + p] > d_delta_old[i] &&
                d_dist_n_k[i * n + p] <= d_delta[i]) {
                int idx = atomicInc((unsigned int *) &d_L_sizes_change[i], n);
                d_L[i * n + idx] = p;
            }
        }
    }

    __syncthreads();
    if (threadIdx.x == 0) {
        d_L_sizes[i] += d_lambda[i] * d_L_sizes_change[i];
    }
}

void gpu_compute_L_keep(int *d_L, int *d_L_sizes_change, int *d_L_sizes, int *d_lambda,
                        float *d_dist_n_k,
                        float *d_delta_old, float *d_delta,
                        int *d_M_current, int *d_M_bad, int num_bad,
                        float *d_data,
                        int n, int d, int k) {
    int number_of_blocks = n / BLOCK_SIZE_SMALL;
    if (n % BLOCK_SIZE_SMALL) number_of_blocks++;
    dim3 grid_k_n(num_bad, number_of_blocks);
    gpu_compute_L_kernel_KEEP_dist << < grid_k_n, min(n, BLOCK_SIZE_SMALL), d * sizeof(float) >> > (d_dist_n_k,
            d_M_current, d_M_bad,
            d_data,
            n, d);

    //compute delta
    gpu_compute_L_kernel_compute_delta_V2 << < 1, k >> > (d_delta, d_dist_n_k, d_M_current, n, k);

    //compute L
    hipMemset(d_L_sizes_change, 0, k * sizeof(int));
    gpu_compute_L_kernel_KEEP_L << < k, min(n, BLOCK_SIZE) >> > (d_lambda, d_L, d_L_sizes, d_L_sizes_change,
            d_dist_n_k,
            d_delta, d_delta_old,
            n);

    hipMemcpy(d_delta_old, d_delta, k * sizeof(float), hipMemcpyDeviceToDevice);
}


__global__
void
gpu_find_dimensions_kernel_KEEP_H(float *d_H,
                                  int *d_L, int *d_L_sizes_change, int *d_lambda,
                                  int *d_M_current,
                                  float *d_data,
                                  int n, int d) {
    int i = blockIdx.x; //independent for different k
    int j = threadIdx.x; //independent for different d

    float sum = 0.;

    int m_i = d_M_current[i];
    int L_i_size_change = d_L_sizes_change[i];
    float data_ij = d_data[m_i * d + j];

    for (int p = blockDim.y * blockIdx.y + threadIdx.y; p < L_i_size_change; p += gridDim.y * blockDim.y) {
        int point = d_L[i * n + p];
        sum += std::abs(d_data[point * d + j] - data_ij);
    }

    atomicAdd(&d_H[i * d + j], d_lambda[i] * sum);
}


__global__
void
gpu_find_dimensions_kernel_KEEP_X(float *d_X, float *d_H, int *d_L_sizes, int d) {

    int i = blockIdx.x; //independent for different k
    int j = threadIdx.x; //independent for different d
    int L_i_size = d_L_sizes[i];

    d_X[i * d + j] = d_H[i * d + j] / L_i_size;
}

void gpu_find_dimensions_keep(int *d_D, float *d_Z, float *d_X, float *d_H,
                              int *d_L, int *d_L_sizes_change, int *d_L_sizes, int *d_lambda,
                              int *d_M_current,
                              float *d_data,
                              int n, int d, int k, int l) {
    int number_of_blocks = (k * d) / BLOCK_SIZE;
    if ((k * d) % BLOCK_SIZE) number_of_blocks++;

    set_all << < number_of_blocks, min(k * d, BLOCK_SIZE) >> > (d_X, 0, k * d);

    int remaining_d = BLOCK_SIZE / d;
    int number_of_blocks_X_join_v2 = (n / k) / remaining_d;
    if ((n / k) % remaining_d) number_of_blocks_X_join_v2++;
    dim3 grid_X_join_v2(k, number_of_blocks_X_join_v2);
    dim3 block_X_join_v2(d, remaining_d);


    gpu_find_dimensions_kernel_KEEP_H << < grid_X_join_v2, block_X_join_v2 >> > (d_H,
            d_L, d_L_sizes_change, d_lambda,
            d_M_current,
            d_data,
            n, d);

    gpu_find_dimensions_kernel_KEEP_X << < k, d >> > (d_X, d_H, d_L_sizes, d);

    gpu_find_dimensions_kernel_Z << < k, d >> > (d_Z, d_X, k, d);


    //compute D
    set_all << < number_of_blocks, min(k * d, BLOCK_SIZE) >> > (d_D, false, k * d);
    dim3 block(min(32, k), min(32, d));
    gpu_find_dimensions_kernel_compute_D << < 1, block, 2 * k * sizeof(float) >> > (d_D, d_Z, k, d, l);
}

__global__
void gpu_replace_medoids_kernel_Keep(int *d_M_bad, int *d_num_bad, int *d_M_current, int *d_M_random, int *d_M,
                                     int *d_M_best,
                                     bool *d_bad, int k) {

    extern __shared__ int s_M_kept[];

    int j = 0;
    for (int i = 0; i < k; i++) {
        if (!d_bad[i]) {
            d_M_current[i] = d_M_best[i];
            s_M_kept[j] = d_M_best[i];
            j += 1;
        }
    }

    int p = 0;
    int l = 0;
    for (int i = 0; i < k; i++) {
        if (d_bad[i]) {
            d_M_bad[l] = i;//todo explain why this is needed in the GPU version
            l++;

            bool is_in = true;
            while (is_in) {
                is_in = false;
                for (int q = 0; q < j; q++) {
                    if (d_M[d_M_random[p]] == s_M_kept[q]) {
                        is_in = true;
                        p++;
                        break;
                    }
                }
            }
            d_M_current[i] = d_M[d_M_random[p]];
            s_M_kept[j] = d_M[d_M_random[p]];
            j++;
            p++;
        }
    }

    d_num_bad[0] = l;
}

__global__
void gpu_replace_medoids_kernel_keep_reset(int *d_L_sizes, float *d_delta_old, float *d_H, int *d_M_bad, int d) {
    int i = d_M_bad[blockIdx.x];
    int j = threadIdx.x;

    d_L_sizes[i] = 0;
    d_delta_old[i] = -1.;
    d_H[i * d + j] = 0.;
}

__global__
void
gpu_compute_L_kernel_sum_dist_SAVE(const int *__restrict__ d_M_idx, const int *__restrict__ d_M,
                                   const float *__restrict__ d_data, float *__restrict__ d_dist_n_Bk,
                                   const bool *__restrict__ d_dist_n_Bk_set,
                                   const int k, const int d, const int n) {
    int l = blockIdx.x;
    int i = d_M_idx[l];
    int m_i = d_M[i];

    extern __shared__ float s_data_i[];

    if (!d_dist_n_Bk_set[i]) {
        if (threadIdx.x < d) {
            s_data_i[threadIdx.x] = d_data[m_i * d + threadIdx.x];
        }

        __syncthreads();

        for (int p = blockIdx.y * blockDim.x + threadIdx.x; p < n; p += gridDim.y * blockDim.x) {//independent
            float sum = 0;
            for (int j = 0; j < d; j++) {//we have plenty to parallelize over here - so we can avoid the atomic
                float sub = d_data[p * d + j] - s_data_i[j];
                sum += sub * sub;
            }
            d_dist_n_Bk[i * n + p] = std::sqrt(sum);
        }
    }
}

__global__
void gpu_compute_L_kernel_sqrt_dist_pre_mark(int *d_M_idx, float *d_dist_n_Bk, bool *d_dist_n_Bk_set, int k, int n) {
    for (int l = threadIdx.x; l < k; l += blockDim.x) {//independent
        int i = d_M_idx[l];
        d_dist_n_Bk_set[i] = true;
    }
}

__global__
void gpu_compute_L_kernel_compute_delta_pre(int *d_M_idx, int *d_M, float *d_delta, float *d_dist, int k, int n) {
    for (int i = blockIdx.x; i < k; i += gridDim.x) {//independent
        d_delta[i] = 1000000.;//todo not nice
        for (int p = threadIdx.x; p < k; p += blockDim.x) {
            if (i != p) {
//                if (d_dist[d_M_idx[i] * n + d_M[d_M_idx[p]]] < d_delta[i]) {//todo be carefull here
//                    d_delta[i] = d_dist[d_M_idx[i] * n + d_M[d_M_idx[p]]];
//                }
                atomicMin(&d_delta[i], d_dist[d_M_idx[i] * n + d_M[d_M_idx[p]]]);
            }
        }
    }
}

__global__
void
gpu_compute_L_kernel_compute_L_pre(int *d_posneg, int *d_M_idx, int *d_L, int *d_L_sizes, int *d_L_sizes_change,
                                   float *d_dist_n_Bk, float *d_delta, float *d_old_delta, int k, int n) {

    for (int i = blockIdx.x; i < k; i += gridDim.x) {//independent
        if (d_old_delta[d_M_idx[i]] > d_delta[i]) {
            if (threadIdx.x == 0)
                d_posneg[i] = -1;
            for (int p = threadIdx.x; p < n; p += blockDim.x) {
                if (d_dist_n_Bk[d_M_idx[i] * n + p] <= d_old_delta[d_M_idx[i]] &&
                    d_dist_n_Bk[d_M_idx[i] * n + p] > d_delta[i]) {
                    int old_size = atomicInc((unsigned int *) &d_L_sizes_change[i], n);
                    d_L[i * n + old_size] = p;
                }
            }
        } else {
            if (threadIdx.x == 0)
                d_posneg[i] = 1;
            for (int p = threadIdx.x; p < n; p += blockDim.x) {
                if (d_dist_n_Bk[d_M_idx[i] * n + p] > d_old_delta[d_M_idx[i]] &&
                    d_dist_n_Bk[d_M_idx[i] * n + p] <= d_delta[i]) {
                    int old_size = atomicInc((unsigned int *) &d_L_sizes_change[i], n);
                    d_L[i * n + old_size] = p;
                }
            }
        }

        __syncthreads();
        if (threadIdx.x == 0) {
            d_L_sizes[d_M_idx[i]] += d_posneg[i] * d_L_sizes_change[i];
        }
    }
}

__global__
void gpu_compute_L_kernel_set_old_delta_pre(int *d_M_idx, float *d_old_delta, float *d_delta, int k) {
    for (int i = threadIdx.x; i < k; i += blockDim.x) {
        d_old_delta[d_M_idx[i]] = d_delta[i];
    }
}

void gpu_compute_L_save(int *d_L, int *d_L_sizes_change, int *d_L_sizes, int *d_lambda,
                        float *d_dist_n_Bk, bool *d_dist_n_Bk_set,
                        float *d_delta_old, float *d_delta,
                        int *d_M, int *d_M_idx,
                        float *d_data,
                        int n, int d, int k) {
    int number_of_blocks = n / BLOCK_SIZE_SMALL;
    if (n % BLOCK_SIZE_SMALL) number_of_blocks++;
    dim3 grid_k_n(k, number_of_blocks);
    gpuErrchk(hipPeekAtLastError());
    gpu_compute_L_kernel_sum_dist_SAVE << < grid_k_n, min(n, BLOCK_SIZE_SMALL), d * sizeof(float) >> > (d_M_idx, d_M,
            d_data,
            d_dist_n_Bk,
            d_dist_n_Bk_set,
            k, d, n);
    gpuErrchk(hipPeekAtLastError());

    gpu_compute_L_kernel_sqrt_dist_pre_mark << < 1, k >> > (d_M_idx, d_dist_n_Bk, d_dist_n_Bk_set, k, n);
    gpuErrchk(hipPeekAtLastError());

    //compute delta
    gpu_compute_L_kernel_compute_delta_pre << < k, k >> > (d_M_idx, d_M, d_delta, d_dist_n_Bk, k, n);
    gpuErrchk(hipPeekAtLastError());

    //compute L
    hipMemset(d_L_sizes_change, 0, k * sizeof(int));
    gpuErrchk(hipPeekAtLastError());
    gpu_compute_L_kernel_compute_L_pre << < k, min(n, BLOCK_SIZE) >> > (d_lambda, d_M_idx, d_L, d_L_sizes,
            d_L_sizes_change,
            d_dist_n_Bk, d_delta, d_delta_old, k, n);
    gpuErrchk(hipPeekAtLastError());

    gpu_compute_L_kernel_set_old_delta_pre << < 1, k >> > (d_M_idx, d_delta_old, d_delta, k);
    gpuErrchk(hipPeekAtLastError());
}


__global__
void
gpu_find_dimensions_kernel_SAVE_H(float *__restrict__ d_X, float *__restrict__ d_H,
                                  const float *__restrict__ d_data,
                                  const int *__restrict__ d_L,
                                  const int *__restrict__ d_L_sizes_change,
                                  const int *__restrict__ d_L_sizes,
                                  const int *__restrict__ d_lambda,
                                  const int *__restrict__ d_M_current,
                                  const int *__restrict__ d_M_idx,
                                  const int k, const int d, const int n) {
    int i = blockIdx.x; //independent for different k
    int j = threadIdx.x; //independent for different d

    float sum = 0.;

    int m_i = d_M_current[i];
    int m_idx = d_M_idx[i];
    int L_i_size_change = d_L_sizes_change[i];
    int posneg_i = d_lambda[i];
    float data_ij = d_data[m_i * d + j];

    for (int p = blockDim.y * blockIdx.y + threadIdx.y; p < L_i_size_change; p += gridDim.y * blockDim.y) {
        int point = d_L[i * n + p];
        sum += std::abs(d_data[point * d + j] - data_ij);
    }

    atomicAdd(&d_H[m_idx * d + j], posneg_i * sum);
}


__global__
void
gpu_find_dimensions_kernel_SAVE_X(float *__restrict__ d_X, float *__restrict__ d_H,
                                  const float *__restrict__ d_data,
                                  const int *__restrict__ d_L,
                                  const int *__restrict__ d_L_sizes_change,
                                  const int *__restrict__ d_L_sizes,
                                  const int *__restrict__ d_lambda,
                                  const int *__restrict__ d_M_current,
                                  const int *__restrict__ d_M_idx,
                                  const int k, const int d, const int n) {

    int i = blockIdx.x; //independent for different k
    int j = threadIdx.x; //independent for different d
    int m_idx = d_M_idx[i];
    int L_i_size = d_L_sizes[m_idx];

    d_X[i * d + j] = d_H[m_idx * d + j] / L_i_size;
}

void gpu_find_dimensions_save(int *d_D, float *d_Z, float *d_X, float *d_H,
                              int *d_L, int *d_L_sizes_change, int *d_L_sizes, int *d_lambda,
                              int *d_M_current, int *d_M_idx,
                              float *d_data,
                              int n, int d, int k, int l) {
    gpuErrchk(hipPeekAtLastError());

    int number_of_blocks = (k * d) / BLOCK_SIZE;
    if ((k * d) % BLOCK_SIZE) number_of_blocks++;
    set_all << < number_of_blocks, min(k * d, BLOCK_SIZE) >> > (d_X, 0, k * d);
    gpuErrchk(hipPeekAtLastError());


    int remaining_d = BLOCK_SIZE / d;
    int number_of_blocks_X_join_v2 = (n / k) / remaining_d;
    if ((n / k) % remaining_d) number_of_blocks_X_join_v2++;
    dim3 grid_X_join_v2(k, number_of_blocks_X_join_v2);
    dim3 block_X_join_v2(d, remaining_d);
    gpuErrchk(hipPeekAtLastError());


    gpu_find_dimensions_kernel_SAVE_H << < grid_X_join_v2, block_X_join_v2 >> > (d_X, d_H, d_data, d_L,
            d_L_sizes_change, d_L_sizes, d_lambda,
            d_M_current, d_M_idx,
            k, d, n);
    gpuErrchk(hipPeekAtLastError());

    gpu_find_dimensions_kernel_SAVE_X << < k, d >> > (d_X, d_H, d_data, d_L,
            d_L_sizes_change, d_L_sizes, d_lambda,
            d_M_current, d_M_idx,
            k, d, n);
    gpuErrchk(hipPeekAtLastError());

    gpu_find_dimensions_kernel_Z << < k, d >> > (d_Z, d_X, k, d);
    gpuErrchk(hipPeekAtLastError());

    //compute D
    set_all << < number_of_blocks, min(k * d, BLOCK_SIZE) >> > (d_D, false, k * d);
    gpuErrchk(hipPeekAtLastError());
    dim3 block(min(32, k), min(32, d));
    gpu_find_dimensions_kernel_compute_D << < 1, block, 2 * k * sizeof(float) >> > (d_D, d_Z, k, d, l);
    gpuErrchk(hipPeekAtLastError());

}

__global__
void
gpu_update_best_kernel_init_k_pre(int *d_M_idx, int *d_M_idx_best, int *d_termination_criterion,
                                  int *d_M_best, int *d_M_current,
                                  bool *d_bad, int k) {

    if (d_termination_criterion[0] == 0) {//todo worng!!!! then we allways pick the last????
        for (int i = threadIdx.x; i < k; i += blockDim.x) {
            d_M_best[i] = d_M_current[i];
            d_M_idx_best[i] = d_M_idx[i];
            d_bad[i] = false;
        }
    }
}

void
gpu_update_best_SAVE(int *d_M_idx, int *d_M_idx_best, float *d_cost,
                     float *d_cost_best,
                     int *d_termination_criterion,
                     int *d_M_best, int *d_M_current,
                     int *d_C, int *d_C_sizes, int *d_C_best, int *d_C_sizes_best,
                     bool *d_bad,
                     float min_deviation, int n, int k) {

    gpu_update_best_kernel_is_best << < 1, 1 >> > (d_cost, d_cost_best, d_termination_criterion);
    gpu_update_best_kernel_init_k_pre << < 1, k >> > (d_M_idx, d_M_idx_best, d_termination_criterion, d_M_best,
            d_M_current, d_bad, k);
    gpu_update_best_kernel_C << < k, BLOCK_SIZE >> >
    (d_C_best, d_C_sizes_best, d_C, d_C_sizes, d_termination_criterion, n);
    gpu_update_best_kernel_find_bad << < 1, k >> >
    (d_C_sizes_best, d_termination_criterion, d_bad, k, n, min_deviation);

}

__global__
void
gpu_replace_medoids_kernel_pre(int *d_M_idx, int *d_M_idx_best, int *d_M_current, int *d_M_random, int *d_M, int Bk,
                               int *d_M_best, bool *d_bad,
                               int k, int n) {

    int j = 0;
    for (int i = 0; i < k; i++) {
        if (!d_bad[i]) {
            d_M_current[j] = d_M_best[i];
            d_M_idx[j] = d_M_idx_best[i];
            j += 1;
        }
    }

    int p = 0;
    while (j < k) {
        bool is_in = false;
        for (int i = 0; i < j; i++) {
            if (d_M[d_M_random[p]] == d_M_current[i]) {
                is_in = true;
                break;
            }
        }
        if (!is_in) {
            d_M_current[j] = d_M[d_M_random[p]];
            d_M_idx[j] = d_M_random[p];
            j += 1;
        }
        p += 1;
    }
}

void
GPU_FAST_PROCLUS(int *h_C, int *h_D, float *h_data, int n, int d, int k, int l, float a, float b, float min_deviation,
                 int termination_rounds) {
    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());

    //getting constants
    l = min(l, d);
    int Ak = min(n, int(a * k));
    int Bk = min(n, int(b * k));

    //copying data to the GPU
    float *d_data = copy_H_to_D(h_data, n * d);
    gpuErrchk(hipPeekAtLastError());

    //initializing random generator for cuda
    hiprandState *d_state;
    hipMalloc(&d_state, BLOCK_SIZE * sizeof(hiprandState));
    init_seed << < 1, BLOCK_SIZE >> > (d_state, 42);
    gpuErrchk(hipPeekAtLastError());

    int *d_state_fixed;
    gpuErrchk(hipPeekAtLastError());

    //initializing cuda arrays
    bool *d_bad = gpu_malloc_bool(k);
    int *d_C = gpu_malloc_int(k * n);
    int *d_C_sizes = gpu_malloc_int(k);
    gpuErrchk(hipPeekAtLastError());
    int *d_C_best = gpu_malloc_int(n * k);
    gpuErrchk(hipPeekAtLastError());
    int *d_C_sizes_best = gpu_malloc_int(k);
    int *d_C_result = gpu_malloc_int(n);
    float *d_cost = gpu_malloc_float(1);
    float *d_cost_best = gpu_malloc_float(1);
    int *d_D = gpu_malloc_int(k * d);
    int *d_Ds = gpu_malloc_int(k * d);
    int *d_D_sizes = gpu_malloc_int(k);
    float *d_delta = gpu_malloc_float(k);
    float *d_delta_old = gpu_malloc_float(Bk);
    gpuErrchk(hipPeekAtLastError());
    float *d_dist_n_Bk = gpu_malloc_float_zero(n * Bk);
    gpuErrchk(hipPeekAtLastError());
    bool *d_dist_n_Bk_set = gpu_malloc_bool_false(Bk);
    float *d_H = gpu_malloc_float_zero(Bk * d);
    int *d_L = gpu_malloc_int(n * k);
    int *d_L_sizes = gpu_malloc_int_zero(Bk);
    int *d_L_sizes_change = gpu_malloc_int(k);
    int *d_lambda = gpu_malloc_int(k);
    int *d_lock = gpu_malloc_int(n);
    int *d_M_best = gpu_malloc_int(k);
    int *d_M_current = gpu_malloc_int(k);
    int *d_M_idx = gpu_malloc_int(k);
    int *d_M_idx_best = gpu_malloc_int(k);
    int *d_M_random = gpu_malloc_int(Bk);
    int *d_S = gpu_malloc_int(n);
    float *d_sigma = gpu_malloc_float(k);
    int *d_termination_criterion = gpu_malloc_int_zero(1);
    float *d_X = gpu_malloc_float(k * d);
    float *d_Z = gpu_malloc_float(k * d);


    //allocate result
    int *d_M = gpu_malloc_int(Bk);

    //allocate tmp
    float *d_dist = gpu_malloc_float(Ak);
    int *d_prev = gpu_malloc_int_zero(1);
    float *d_max_value = gpu_malloc_float_zero(1);

    gpuErrchk(hipPeekAtLastError());

    //// Initialization Phase ////
    fill_with_indices(d_S, n);
    gpuErrchk(hipPeekAtLastError());

    gpu_random_sample_locked(d_S, Ak, n, d_state, d_lock);

    gpu_greedy(d_data, d_S,
               d_M, d_dist, d_prev, d_max_value,
               Bk, Ak, d, n);
    gpuErrchk(hipPeekAtLastError());

    //free tmp
    hipFree(d_dist);
    hipFree(d_prev);
    hipFree(d_max_value);

    //// Iterative Phase ///
    fill_with_indices(d_M_random, Bk);
    gpu_random_sample_locked(d_M_random, k, Bk, d_state, d_lock);
    gpu_gather_1d(d_M_current, d_M, d_M_random, k);
    hipMemcpy(d_M_best, d_M_current, k * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(d_M_idx, d_M_random, k * sizeof(int), hipMemcpyDeviceToDevice);
    hipMemcpy(d_M_idx_best, d_M_random, k * sizeof(int), hipMemcpyDeviceToDevice);
    gpuErrchk(hipPeekAtLastError());

    int termination_criterion = 0;
    set(d_cost_best, 0, 1000000.);

    int number_of_blocks = Bk / BLOCK_SIZE;
    if (Bk % BLOCK_SIZE) number_of_blocks++;
    set_all << < number_of_blocks, min(Bk, BLOCK_SIZE) >> > (d_delta_old, -1., Bk);
    gpuErrchk(hipPeekAtLastError());

    while (termination_criterion < termination_rounds) {

        //// compute L ////
        gpu_compute_L_save(d_L, d_L_sizes_change, d_L_sizes, d_lambda,
                           d_dist_n_Bk, d_dist_n_Bk_set,
                           d_delta_old, d_delta,
                           d_M, d_M_idx,
                           d_data,
                           n, d, k);
        gpuErrchk(hipPeekAtLastError());

        //// find dimensions ////
        gpu_find_dimensions_save(d_D, d_Z, d_X, d_H,
                                 d_L, d_L_sizes_change, d_L_sizes, d_lambda,
                                 d_M_current, d_M_idx,
                                 d_data,
                                 n, d, k, l);
        gpuErrchk(hipPeekAtLastError());

        //// assign points /////
        gpu_assign_points(d_C, d_C_sizes,
                          d_D, d_Ds, d_D_sizes,
                          d_M_current,
                          d_data,
                          n, d, k);
        gpuErrchk(hipPeekAtLastError());

        //// evaluate clustering ////
        gpu_evaluate_cluster(d_cost,
                             d_C, d_C_sizes,
                             d_D, d_D_sizes,
                             d_data,
                             n, d, k);
        gpuErrchk(hipPeekAtLastError());

        //// update best ////
        termination_criterion += 1;
        gpu_update_best_SAVE(d_M_idx, d_M_idx_best, d_cost, d_cost_best,
                             d_termination_criterion,
                             d_M_best, d_M_current,
                             d_C, d_C_sizes, d_C_best, d_C_sizes_best,
                             d_bad,
                             min_deviation, n, k);
        gpuErrchk(hipPeekAtLastError());

        if (termination_criterion >= termination_rounds) {
            //only read from device version of termination_criterion as few times as possible
            hipMemcpy(&termination_criterion, d_termination_criterion, sizeof(int), hipMemcpyDeviceToHost);
        }

        //replace bad medoids
        gpu_random_sample_locked(d_M_random, k, Bk, d_state, d_lock);
        gpu_replace_medoids_kernel_pre << < 1, 1 >> > (d_M_idx, d_M_idx_best, d_M_current, d_M_random, d_M, Bk,
                d_M_best, d_bad, k, n);
        gpuErrchk(hipPeekAtLastError());

    }

    //// Refinement Phase ////
    gpu_find_dimensions(d_D, d_Z, d_X,
                        d_C_best, d_C_sizes_best,
                        d_M_best,
                        d_data,
                        n, d, k, l);
    gpuErrchk(hipPeekAtLastError());

    gpu_assign_points(d_C_best, d_C_sizes_best,
                      d_D, d_Ds, d_D_sizes,
                      d_M_best,
                      d_data,
                      n, d, k);
    gpuErrchk(hipPeekAtLastError());

    remove_outliers(d_C_result, d_C_best, d_C_sizes_best,
                    d_D,
                    d_delta,
                    d_M_best,
                    d_data,
                    n, d, k);
    gpuErrchk(hipPeekAtLastError());

    // building result

    hipMemcpy(h_D, d_D, k * d * sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(h_C, d_C_result, n * sizeof(int), hipMemcpyDeviceToHost);

    gpuErrchk(hipPeekAtLastError());

    // free all
    hipFree(d_bad);
    hipFree(d_C);
    hipFree(d_C_sizes);
    hipFree(d_C_best);
    hipFree(d_C_sizes_best);
    hipFree(d_C_result);
    hipFree(d_cost);
    hipFree(d_cost_best);
    hipFree(d_D);
    hipFree(d_Ds);
    hipFree(d_D_sizes);
    hipFree(d_data);
    hipFree(d_delta);
    hipFree(d_delta_old);
    hipFree(d_dist_n_Bk);
    hipFree(d_dist_n_Bk_set);
    hipFree(d_H);
    hipFree(d_L);
    hipFree(d_L_sizes);
    hipFree(d_L_sizes_change);
    hipFree(d_lambda);
    hipFree(d_lock);
    hipFree(d_M);
    hipFree(d_M_best);
    hipFree(d_M_current);
    hipFree(d_M_idx);
    hipFree(d_M_idx_best);
    hipFree(d_M_random);
    hipFree(d_S);
    hipFree(d_sigma);
    hipFree(d_state);
    hipFree(d_termination_criterion);
    hipFree(d_X);
    hipFree(d_Z);

    hipDeviceSynchronize();
    gpuErrchk(hipPeekAtLastError());
}